#include "hip/hip_runtime.h"
// #include <torch/extension.h>
// #include <ATen/cuda/HIPContext.h>
// #include <c10/cuda/CUDACachingAllocator.h>

#pragma once

#include <optional>
#include <algorithm>

#include "tensorrt_llm/kernels/mixtureOfExperts/moe_kernels.h"
#include "tensorrt_llm/kernels/cutlass_kernels/cutlass_preprocessors.h"
#include "utils.h"
#include <cstdio>

template <typename T>
void print_gpu_data(T* gpu_data, size_t num_elements) {
    // 在主机上创建一个缓冲区来接收 GPU 数据
    float* host_data = new float[num_elements];

    // 创建一个用于转换数据的缓冲区
    T* temp_data = new T[num_elements];

    // 从 GPU 拷贝数据到临时缓冲区（float32）
    hipMemcpy(temp_data, gpu_data, sizeof(float) * num_elements, hipMemcpyDeviceToHost);

    // 将转换后的数据拷贝到主机数据缓冲区
    for (size_t i = 0; i < num_elements; i++) {
        host_data[i] = static_cast<float>(temp_data[i]);
    }

    // 打印前几个元素
    for (size_t i = 0; i < num_elements; i++) {
        printf("gpu_data[%zu] = %f\n", i, host_data[i]);
    }

    // 释放内存
    delete[] host_data;
    delete[] temp_data;
}




using paddle::Tensor;

int getSMVersion() {
    int device = -1;
    hipGetDevice(&device);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    return props.major * 10 + props.minor;
}


template<typename T, typename WeightType, typename OutputType = T>
std::vector<tensorrt_llm::cutlass_extensions::CutlassGemmConfig> getFilteredConfigs(
    tensorrt_llm::kernels::CutlassMoeFCRunner<T, WeightType, OutputType>& moe_runner, int sm) {
    auto tactics = moe_runner.getTactics();
    if (sm == 89) {
        // Filter some unsupported configs for L40S
        auto it = std::remove_if(tactics.begin(), tactics.end(),
            [&](auto conf) {
                using tensorrt_llm::cutlass_extensions::CutlassTileConfig;
                auto checks = std::vector{
                    // Fail for BF16/FP16
                    conf.tile_config == CutlassTileConfig::CtaShape128x128x64_WarpShape64x32x64,
                    conf.tile_config == CutlassTileConfig::CtaShape64x128x64_WarpShape32x64x64 && conf.stages == 4,
                    // Fail for FP8
                    false && conf.tile_config == CutlassTileConfig::CtaShape16x256x128_WarpShape16x64x128
                        && conf.stages >= 3,
                };

                return std::any_of(checks.begin(), checks.end(), [](auto v) { return v; });
            });
        tactics.erase(it, tactics.end());
    }

    if (tactics.empty()) {
        throw std::runtime_error("No valid GEMM tactics found");
    }

    return tactics;
}


// 第三个模版参数默认是T
template<typename T, typename WeightType, typename OutputType = T>
std::pair<tensorrt_llm::cutlass_extensions::CutlassGemmConfig, tensorrt_llm::cutlass_extensions::CutlassGemmConfig> 
selectTacticsForArch(tensorrt_llm::kernels::CutlassMoeFCRunner<T, WeightType, OutputType>& moe_runner, int sm) {
    bool is_sm90 = sm >= 90;
    auto tactics = getFilteredConfigs(moe_runner, sm);
    auto it = std::find_if(tactics.begin(), tactics.end(), [is_sm90](auto& c) { return c.is_sm90 == is_sm90; });
    if (it == tactics.end()) {
        // Fall back to any tactic
        std::cout << "WARNING: Could not find config for sm version " << sm << std::endl;
        return std::make_pair(tactics[0], tactics[0]);
    }

    return std::make_pair(*it, *it);
}



tensorrt_llm::ActivationType getActivationType(std::string activation_type_str)
{
    if (activation_type_str == "Gelu" || activation_type_str == "gelu") {
        return tensorrt_llm::ActivationType::Gelu;
    }
    else if (activation_type_str == "Relu" || activation_type_str == "relu") {
        return tensorrt_llm::ActivationType::Relu;
    }
    else if (activation_type_str == "Silu" || activation_type_str == "silu") {
        return tensorrt_llm::ActivationType::Silu;
    }
    else if (activation_type_str == "GeGLU" || activation_type_str == "geglu" || activation_type_str == "gated-gelu") {
        return tensorrt_llm::ActivationType::Geglu;
    }
    else if (activation_type_str == "Swiglu") {
        return tensorrt_llm::ActivationType::Swiglu;
    }
    else {
        std::cout << "Activation Type: " <<  activation_type_str << " not supported !";
    }
    return tensorrt_llm::ActivationType::InvalidType;
}


template<typename T, typename WeightType>
Tensor trt_llm_fused_moe_helper(Tensor input_activations, 
                                 Tensor gating_output, 
                                 Tensor fc1_expert_weights, 
                                 tensorrt_llm::ActivationType fc1_activation_type,
                                 Tensor fc2_expert_weights, 
                                 const int active_rows, 
                                 const int k,
                                 paddle::optional<paddle::Tensor> scale1 = nullptr,
                                 paddle::optional<paddle::Tensor> scale2 = nullptr,
                                 paddle::optional<paddle::Tensor> scale3 = nullptr,
                                 const std::string& quant_method = "none")
{
    typedef DataTypeMapper<T> traits_t;
    typedef typename traits_t::DataType DataType_;
    typedef typename traits_t::data_t data_t;

    typedef DataTypeMapper<WeightType> traits_w;
    typedef typename traits_w::DataType DataType_w;
    typedef typename traits_w::data_t data_w;

    const int num_rows = input_activations.shape()[0];
    const int hidden_size = input_activations.shape()[1];
    const int inter_size = fc2_expert_weights.shape()[1];
    const int num_experts = gating_output.shape()[0];
    auto stream = input_activations.stream();
    auto place = input_activations.place();

    data_t* input_act_ptr = get_ptr<data_t>(input_activations);
    float* gating_output_ptr = get_ptr<float>(gating_output);

    float* scale1_ptr = scale1 ? get_ptr<float>(scale1) : nullptr;
    float* scale2_ptr = scale2 ? get_ptr<float>(scale2) : nullptr;
    float* scale3_ptr = scale3 ? get_ptr<float>(scale3) : nullptr;

    data_t* fc1_expert_biases_ptr = nullptr;
    data_t* fc2_expert_biases_ptr = nullptr;


    bool* finished_ptr = nullptr;

    tensorrt_llm::kernels::MOEParallelismConfig moe_parallel_config = tensorrt_llm::kernels::MOEParallelismConfig(1, 0, 1, 0);

    
    // 根据启用的量化方法设置量化参数
    tensorrt_llm::kernels::QuantParams quant_params;
    if (quant_method == "fp8_pre_tensor") {
        std::cout <<"fp8_pre_tensor" << std::endl;
        quant_params = tensorrt_llm::kernels::QuantParams::FP8(scale1_ptr, scale2_ptr, scale3_ptr);
    } else if (quant_method == "weight_only_int8") {
        quant_params = tensorrt_llm::kernels::QuantParams::Int(scale1_ptr, scale2_ptr);
        std::cout <<   "weight_only_int8 quant parmra done !" << std::endl;
    } else if (quant_method == "weight_only_int4") {
        quant_params = tensorrt_llm::kernels::QuantParams::Int(scale1_ptr, scale2_ptr);
    }


    int sm = getSMVersion();
    tensorrt_llm::kernels::CutlassMoeFCRunner<T, WeightType> moe_runner;

    auto [tactic1, tactic2] = selectTacticsForArch(moe_runner, sm);
    moe_runner.setTactic(std::make_optional(tactic1), std::make_optional(tactic2));

    auto bytes = moe_runner.getWorkspaceSize(num_rows, hidden_size, inter_size, num_experts, k, fc1_activation_type, 
                                             tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE, moe_parallel_config);

    auto workspace_tensor = paddle::empty({static_cast<int>(bytes)}, paddle::DataType::UINT8, place);
    uint8_t* uint8_ptr = get_ptr<uint8_t>(workspace_tensor);
    char* workspace_ptr = reinterpret_cast<char*>(uint8_ptr);

    auto fc2_output = paddle::empty({k * num_rows, hidden_size}, input_activations.dtype(), place);
    auto expert_scales = paddle::empty({num_rows, k}, input_activations.dtype(), place);
    data_t* expert_scales_ptr = get_ptr<data_t>(expert_scales);

    auto expanded_source_row_to_expanded_dest_row = paddle::empty({num_rows, k}, paddle::DataType::INT32, place);
    int* expanded_source_row_to_expanded_dest_row_ptr = get_ptr<int>(expanded_source_row_to_expanded_dest_row);

    auto expert_for_source_row = paddle::empty({num_rows, k}, paddle::DataType::INT32, place);
    int* expert_for_source_row_ptr = get_ptr<int>(expert_for_source_row);

    auto output_tensor = paddle::empty({num_rows, hidden_size}, input_activations.dtype(), place);
    data_t* output_tensor_ptr = get_ptr<data_t>(output_tensor);



    if (quant_method == "weight_only_int8") {
        std::cout << "start runMoe 99999"<< std::endl;
        // auto w1 = reinterpret_cast<uint8_t*>(fc1_expert_weights.data<uint8_t>());
        // auto w2 = reinterpret_cast<uint8_t*>(fc2_expert_weights.data<uint8_t>());
        // printf("fc1_expert_weights = %f\n", w1[0]);
        // printf("fc1_expert_weights = %f\n", w2[0]);
        // print_gpu_data<uint8_t>(w1, 10);
        // print_gpu_data<uint8_t>(w2, 10);
        // std::cout << "print end "<< std::endl;
        moe_runner.runMoe(input_act_ptr,
                      gating_output_ptr,
                      reinterpret_cast<uint8_t*>(fc1_expert_weights.data<int8_t>()),
                      fc1_expert_biases_ptr,
                      fc1_activation_type,
                      reinterpret_cast<uint8_t*>(fc1_expert_weights.data<int8_t>()),
                      fc2_expert_biases_ptr,
                      quant_params,
                      num_rows,
                      hidden_size,
                      inter_size,
                      num_experts,
                      k,
                      workspace_ptr,
                      output_tensor_ptr,
                      finished_ptr,
                      active_rows,
                      expert_scales_ptr,
                      expanded_source_row_to_expanded_dest_row_ptr,
                      expert_for_source_row_ptr,
                      0.2f,  // sparse_mixer_epsilon
                      moe_parallel_config,
                      tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE,
                      stream);

    } else if (quant_method == "weight_only_int4") {
        std::cout << "start runMoe 4444"<< std::endl;
        moe_runner.runMoe(input_act_ptr,
                      gating_output_ptr,
                      reinterpret_cast<cutlass::uint4b_t*>(fc1_expert_weights.data<int8_t>()),
                      fc1_expert_biases_ptr,
                      fc1_activation_type,
                      reinterpret_cast<cutlass::uint4b_t*>(fc2_expert_weights.data<int8_t>()),
                      fc2_expert_biases_ptr,
                      quant_params,
                      num_rows,
                      hidden_size,
                      inter_size,
                      num_experts,
                      k,
                      workspace_ptr,
                      output_tensor_ptr,
                      finished_ptr,
                      active_rows,
                      expert_scales_ptr,
                      expanded_source_row_to_expanded_dest_row_ptr,
                      expert_for_source_row_ptr,
                      0.2f,  // sparse_mixer_epsilon
                      moe_parallel_config,
                      tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE,
                      stream);
    } else {
        std::cout << "start runMoe"<< std::endl;
        // auto hah = get_ptr<data_w>(fc1_expert_weights);
        // print_gpu_data<data_w>(hah, 10);
        moe_runner.runMoe(input_act_ptr,
                      gating_output_ptr,
                      get_ptr<data_w>(fc1_expert_weights),
                      fc1_expert_biases_ptr,
                      fc1_activation_type,
                      get_ptr<data_w>(fc2_expert_weights),
                      fc2_expert_biases_ptr,
                      quant_params,
                      num_rows,
                      hidden_size,
                      inter_size,
                      num_experts,
                      k,
                      workspace_ptr,
                      output_tensor_ptr,
                      finished_ptr,
                      active_rows,
                      expert_scales_ptr,
                      expanded_source_row_to_expanded_dest_row_ptr,
                      expert_for_source_row_ptr,
                      0.2f,  // sparse_mixer_epsilon
                      moe_parallel_config,
                      tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE,
                      stream);
    }

    
    return output_tensor;
}


template<typename T, typename WeightType>
Tensor trt_llm_fused_moe_helper_fp8_per_tensor(Tensor input_activations, 
                                 Tensor gating_output, 
                                 Tensor fc1_expert_weights, 
                                 tensorrt_llm::ActivationType fc1_activation_type,
                                 Tensor fc2_expert_weights, 
                                 const int active_rows, 
                                 const int k,
                                 paddle::optional<paddle::Tensor> scale1 = nullptr,
                                 paddle::optional<paddle::Tensor> scale2 = nullptr,
                                 paddle::optional<paddle::Tensor> scale3 = nullptr,
                                 const std::string& quant_method = "none")
{
    typedef DataTypeMapper<T> traits_t;
    typedef typename traits_t::DataType DataType_;
    typedef typename traits_t::data_t data_t;

    typedef DataTypeMapper<WeightType> traits_w;
    typedef typename traits_w::DataType DataType_w;
    typedef typename traits_w::data_t data_w;

    const int num_rows = input_activations.shape()[0];
    const int hidden_size = input_activations.shape()[1];
    const int inter_size = fc2_expert_weights.shape()[1];
    const int num_experts = gating_output.shape()[0];
    auto stream = input_activations.stream();
    auto place = input_activations.place();

    data_t* input_act_ptr = get_ptr<data_t>(input_activations);
    float* gating_output_ptr = get_ptr<float>(gating_output);

    float* scale1_ptr = scale1 ? get_ptr<float>(scale1) : nullptr;
    float* scale2_ptr = scale2 ? get_ptr<float>(scale2) : nullptr;
    float* scale3_ptr = scale3 ? get_ptr<float>(scale3) : nullptr;

    data_w* fc1_expert_weights_ptr = get_ptr<data_w>(fc1_expert_weights);
    data_t* fc1_expert_biases_ptr = nullptr;

    data_w* fc2_expert_weights_ptr = get_ptr<data_w>(fc2_expert_weights);
    data_t* fc2_expert_biases_ptr = nullptr;

    bool* finished_ptr = nullptr;

    tensorrt_llm::kernels::MOEParallelismConfig moe_parallel_config = tensorrt_llm::kernels::MOEParallelismConfig(1, 0, 1, 0);

    // 根据启用的量化方法设置量化参数
    tensorrt_llm::kernels::QuantParams quant_params;
    quant_params = tensorrt_llm::kernels::QuantParams::FP8(scale1_ptr, scale2_ptr, scale3_ptr);

    int sm = getSMVersion();
    tensorrt_llm::kernels::CutlassMoeFCRunner<T, WeightType, __hip_bfloat16> moe_runner;


    auto [tactic1, tactic2] = selectTacticsForArch(moe_runner, sm);
    moe_runner.setTactic(std::make_optional(tactic1), std::make_optional(tactic2));

    auto bytes = moe_runner.getWorkspaceSize(num_rows, hidden_size, inter_size, num_experts, k, fc1_activation_type, 
                                             tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE, moe_parallel_config);

    auto workspace_tensor = paddle::empty({static_cast<int>(bytes)}, paddle::DataType::UINT8, place);
    uint8_t* uint8_ptr = get_ptr<uint8_t>(workspace_tensor);
    char* workspace_ptr = reinterpret_cast<char*>(uint8_ptr);

    auto fc2_output = paddle::empty({k * num_rows, hidden_size}, input_activations.dtype(), place);
    auto expert_scales = paddle::empty({num_rows, k}, input_activations.dtype(), place);
    data_t* expert_scales_ptr = get_ptr<data_t>(expert_scales);

    auto expanded_source_row_to_expanded_dest_row = paddle::empty({num_rows, k}, paddle::DataType::INT32, place);
    int* expanded_source_row_to_expanded_dest_row_ptr = get_ptr<int>(expanded_source_row_to_expanded_dest_row);

    auto expert_for_source_row = paddle::empty({num_rows, k}, paddle::DataType::INT32, place);
    int* expert_for_source_row_ptr = get_ptr<int>(expert_for_source_row);

    auto output_tensor = paddle::empty({num_rows, hidden_size}, input_activations.dtype(), place);
    data_t* output_tensor_ptr = get_ptr<data_t>(output_tensor);

    moe_runner.runMoe(input_act_ptr,
                      gating_output_ptr,
                      fc1_expert_weights_ptr,
                      fc1_expert_biases_ptr,
                      fc1_activation_type,
                      fc2_expert_weights_ptr,
                      fc2_expert_biases_ptr,
                      quant_params,
                      num_rows,
                      hidden_size,
                      inter_size,
                      num_experts,
                      k,
                      workspace_ptr,
                      output_tensor_ptr,
                      finished_ptr,
                      active_rows,
                      expert_scales_ptr,
                      expanded_source_row_to_expanded_dest_row_ptr,
                      expert_for_source_row_ptr,
                      0.2f,  // sparse_mixer_epsilon
                      moe_parallel_config,
                      tensorrt_llm::kernels::MOEExpertScaleNormalizationMode::RENORMALIZE,
                      stream);

    return output_tensor;
}


std::vector<paddle::Tensor> TrtLLMFusedMoe(const paddle::Tensor&     input_activations, //(num_tokens, hidden_size)
                const paddle::Tensor&      gating_output, //(num_tokens, num_experts)
                const paddle::Tensor&      fc1_expert_weights, //(num_experts, hidden_size, inter_size * 2)
                const paddle::Tensor&      fc2_expert_weights, //(num_experts, inter_size, hidden_size)
                const paddle::optional<paddle::Tensor>& scale1,
                const paddle::optional<paddle::Tensor>& scale2,
                const paddle::optional<paddle::Tensor>& scale3,
                const std::string& fc1_activation_type_str,
                int     active_rows,
                int     k,
                const std::string& quant_method="none")
{

    const auto _st = input_activations.dtype();
    const auto weight_type = fc1_expert_weights.dtype();

    const int num_rows    = input_activations.shape()[0];
    const int hidden_size = input_activations.shape()[1];
    const int num_experts = gating_output.shape()[0];

    const auto quant_type = fc2_expert_weights.dtype();

    Tensor output_tensor;

    tensorrt_llm::ActivationType fc1_activation_type = tensorrt_llm::ActivationType::InvalidType;
    if (fc1_activation_type_str == "identity") {
        fc1_activation_type = tensorrt_llm::ActivationType::Identity;
    }
    else {
        fc1_activation_type = getActivationType(fc1_activation_type_str);
    }

    std::cout << "start ! "<< std::endl;
    std::cout<< quant_method  << std::endl;
    switch (_st) {
         case paddle::DataType::FLOAT32: {
            std::cout << "1 "<< std::endl;
            if (quant_type == _st) {
                output_tensor = trt_llm_fused_moe_helper<float, float>(input_activations,
                                                                gating_output,
                                                                fc1_expert_weights,
                                                                fc1_activation_type,
                                                                fc2_expert_weights,
                                                                active_rows,
                                                                k);
            }
            else {
                std::string err_msg = "Unsupported weight type ";
                throw std::runtime_error(err_msg);
            }
            break;
        }
        case paddle::DataType::FLOAT16: {
            std::cout << "2 "<< std::endl;
            if (quant_type == _st) {
                output_tensor = trt_llm_fused_moe_helper<half, half>(input_activations,
                                                                    gating_output,
                                                                    fc1_expert_weights,
                                                                    fc1_activation_type,
                                                                    fc2_expert_weights,
                                                                    active_rows,
                                                                    k);
            }
            else {
                std::string err_msg = "Unsupported weight type ";
                throw std::runtime_error(err_msg);
            }
            break;
        }
        case paddle::DataType::BFLOAT16: {
            std::cout << "3 "<< std::endl;
            if (quant_type == _st) {
                output_tensor = trt_llm_fused_moe_helper<__hip_bfloat16, __hip_bfloat16>(input_activations,
                                                                                gating_output,
                                                                                fc1_expert_weights,
                                                                                fc1_activation_type,
                                                                                fc2_expert_weights,
                                                                                active_rows,
                                                                                k);
            }
            else {
                if (quant_method == "weight_only_int8") {
                    output_tensor = trt_llm_fused_moe_helper<__hip_bfloat16, uint8_t>(input_activations,
                                                                                gating_output,
                                                                                fc1_expert_weights,
                                                                                fc1_activation_type,
                                                                                fc2_expert_weights,
                                                                                active_rows,
                                                                                k,
                                                                                scale1,
                                                                                scale2,
                                                                                scale3, //scale3不需要
                                                                                quant_method);
                } else if (quant_method == "weight_only_int4") {
                    output_tensor = trt_llm_fused_moe_helper<__hip_bfloat16, cutlass::uint4b_t>(input_activations,
                                                                                gating_output,
                                                                                fc1_expert_weights,
                                                                                fc1_activation_type,
                                                                                fc2_expert_weights,
                                                                                active_rows,
                                                                                k,
                                                                                scale1,
                                                                                scale2,
                                                                                nullptr, //scale3不需要
                                                                                quant_method);
                } else if (quant_method == "fp8_block_wise") {
                    std::string err_msg = "Unsupported weight type ";
                    throw std::runtime_error(err_msg);

                } else {
                    std::string err_msg = "Unsupported weight type ";
                    throw std::runtime_error(err_msg);
                }
            }
            break;
        }
        case paddle::DataType::FLOAT8_E4M3FN: {
            std::cout << "4 "<< std::endl;
            if (quant_type == _st) {
                output_tensor = trt_llm_fused_moe_helper_fp8_per_tensor<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz>(input_activations,
                                                                                gating_output,
                                                                                fc1_expert_weights,
                                                                                fc1_activation_type,
                                                                                fc2_expert_weights,
                                                                                active_rows,
                                                                                k,
                                                                                scale1,
                                                                                scale2,
                                                                                scale3,
                                                                                quant_method);
            }
            else {
                std::string err_msg = "Unsupported weight type ";
                throw std::runtime_error(err_msg);
            }
            break;
        }
        
        default:
            throw std::runtime_error("Wrong Tensor type.");
    }
    return {output_tensor};
}



PD_BUILD_OP(trt_llm_fused_moe)
    .Inputs({"input_activations", "gating_output", "fc1_expert_weights", "fc2_expert_weights", paddle::Optional("scale1"), paddle::Optional("scale2"), paddle::Optional("scale3"),})
    .Outputs({"output_tensor"})
    .Attrs({"fc1_activation_type_str: std::string", "active_rows: int", "k: int", "quant_method:std::string"})
    .SetKernelFn(PD_KERNEL(TrtLLMFusedMoe));