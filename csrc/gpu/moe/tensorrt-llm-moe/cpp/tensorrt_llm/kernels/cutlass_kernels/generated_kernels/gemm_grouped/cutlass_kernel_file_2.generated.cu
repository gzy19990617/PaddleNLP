#include "hip/hip_runtime.h"
#include "tensorrt_llm/kernels/cutlass_kernels/moe_gemm/launchers/moe_gemm_launcher_sm90.inl"
namespace tensorrt_llm
{
namespace kernels
{
namespace cutlass_kernels
{


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<half, half, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_bfloat16, __hip_bfloat16, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<64>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<float, float, float,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<32>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::NONE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<1>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<1>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, half,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


        template void sm90_generic_moe_gemm_kernelLauncher<__hip_fp8_e4m3_fnuz, __hip_fp8_e4m3_fnuz, __hip_bfloat16,
                tensorrt_llm::cutlass_extensions::EpilogueOpDefault, tensorrt_llm::HopperGroupedGemmInput::EpilogueFusion::FINALIZE, cute::Shape<cute::Int<256>, cute::Int<128>, cute::Int<128>>, cute::Shape<cute::Int<2>, cute::Int<2>, cute::Int<1>>, false>
                (HopperGroupedGemmInput, int, int, hipStream_t, int*, size_t*);


} // namespace cutlass_kernels
} // namespace kernels
} // namespace tensorrt_llm
